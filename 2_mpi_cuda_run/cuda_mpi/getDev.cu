#include "hip/hip_runtime.h"
#include "getDev.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void vector_add(float *out, float *a, float *b) {
    out[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void CudaMod::get_dev() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  printf("Number of CUDA devices: %d\n", nDevices); 
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("GPU ID: %d\n", prop.pciBusID);
  }


  long int N = 10000000;
  float *a, *b, *out; 

  // Allocate memory
  a   = (float*)malloc(sizeof(float) * N);
  b   = (float*)malloc(sizeof(float) * N);
  out = (float*)malloc(sizeof(float) * N);

  // Initialize array
  for(int i = 0; i < N; i++){
    a[i] = 1.0f; b[i] = 2.0f;
  }


  printf("Executing vector add\n");

  vector_add<<<N,8>>>(out, a, b);
  free(a);
  free(b);
  free(out);
}

